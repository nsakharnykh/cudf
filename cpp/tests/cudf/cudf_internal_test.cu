/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "gtest/gtest.h"
#include <hip/hip_runtime_api.h>
#include <cstring>

#include <cudf.h>
#include <rmm/rmm.h>

// If this test fails, it means an error code was added without
// adding support to gdf_error_get_name().
TEST(GdfInternalTest, NameEveryError) {
	for (int i = 0; i < N_GDF_ERRORS; i++)
    {
        const char *res = gdf_error_get_name((gdf_error)i);
        ASSERT_EQ(0, strstr(res, "Unknown error"));
    }
}
